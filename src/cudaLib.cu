#include "hip/hip_runtime.h"

#include "cudaLib.cuh"

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ 
void saxpy_gpu (float* x, float* y, float scale, int size) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size) y[i] = scale * x[i] + y[i];
}

int runGpuSaxpy(int vectorSize) {
	std::cout << "Hello GPU Saxpy!\n";

	int i;

	float scale = 6.3f;
	float * x = (float *)malloc(vectorSize * sizeof(float));
	float * y = (float *)malloc(vectorSize * sizeof(float));
	float * result = (float *)malloc(vectorSize * sizeof(float));

	if(!x || !y || !result) {
		std::cout << "Malloc failed";
		return -1;
	}

	// generate random vectors
	for(i = 0; i < vectorSize; i ++) {
		x[i] = (float)(rand() % 1000);
		y[i] = (float)(rand() % 1000);
	}

	// assemble and launch gpu kernel
	float * gpu_x;
	hipMalloc(&gpu_x, vectorSize * sizeof(float));
	float * gpu_y;
	hipMalloc(&gpu_y, vectorSize * sizeof(float));

	hipMemcpy(gpu_x, x, vectorSize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(gpu_y, y, vectorSize * sizeof(float), hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocks = (vectorSize + threadsPerBlock - 1) / threadsPerBlock;
	saxpy_gpu<<<blocks, threadsPerBlock>>>(gpu_x, gpu_y, scale, vectorSize);

	hipMemcpy(result, gpu_y, vectorSize * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(gpu_x);
	hipFree(gpu_y);

	// check result
	float fudgeFactor = 0.001;
	int error_count = 0;
	for(i = 0; i < vectorSize; i ++) {
		float exp_result = scale * x[i] + y[i];
		if(
			result[i] < exp_result - fudgeFactor ||
			result[i] > exp_result + fudgeFactor
		) {
			if(error_count < 20) std::cout << "Got: " << result[i] << " Expected: " << exp_result << "\n";
			error_count ++;
		}
	}

	std::cout << "Found " << error_count << " / " << vectorSize << " errors \n";

	return 0;
}

/* 
 Some helpful definitions

 generateThreadCount is the number of threads spawned initially. Each thread is responsible for sampleSize points. 
 *pSums is a pointer to an array that holds the number of 'hit' points for each thread. The length of this array is pSumSize.

 reduceThreadCount is the number of threads used to reduce the partial sums.
 *totals is a pointer to an array that holds reduced values.
 reduceSize is the number of partial sums that each reduceThreadCount reduces.

*/

#include <hiprand/hiprand_kernel.h>

// generate n random points and calculate the hits
__global__
void generatePoints (uint64_t * pSums, uint64_t pSumSize, uint64_t sampleSize) {
	int thr_i = blockDim.x * blockIdx.x + threadIdx.x;
	if(thr_i >= pSumSize) return;
	int i;
	
	hiprandState_t rng;
	hiprand_init(clock64(), thr_i, 0, &rng);

	pSums[thr_i] = 0; // zero hits so far
	for(i = 0; i < sampleSize; i ++) {
		float x, y;
		x = hiprand_uniform(&rng); y = hiprand_uniform(&rng);
		if(x*x + y*y <= 1.0f) pSums[thr_i] ++;
	}
}

// reduce the sums into the subtotals
__global__ 
void reduceCounts (uint64_t * pSums, uint64_t * totals, uint64_t pSumSize, uint64_t reduceSize) {
	int thr_i = blockDim.x * blockIdx.x + threadIdx.x;
	int i;

	totals[thr_i] = 0;
	for(i = 0; i < reduceSize; i ++) 
		if(i + thr_i * reduceSize < pSumSize) totals[thr_i] += pSums[i + thr_i * reduceSize];
}


int runGpuMCPi (uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {

	//  Check CUDA device presence
	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		std::cout << "CUDA device missing!\n";
		return -1;
	}

	auto tStart = std::chrono::high_resolution_clock::now();
		
	float approxPi = estimatePi(generateThreadCount, sampleSize, 
		reduceThreadCount, reduceSize);
	
	std::cout << "Estimated Pi = " << approxPi << "\n";

	auto tEnd= std::chrono::high_resolution_clock::now();

	std::chrono::duration<double> time_span = (tEnd- tStart);
	std::cout << "It took " << time_span.count() << " seconds.";

	return 0;
}

double estimatePi(uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {
	int i;

	double approxPi = 0;

	uint64_t * totals = (uint64_t *)malloc(reduceThreadCount * sizeof(uint64_t));
	uint64_t * sums = (uint64_t *)malloc(generateThreadCount * sizeof(uint64_t));

	if(!totals || !sums) {
		std::cout << "Malloc failed";
		return -1;
	}

	// assemble and launch generate kernel
	uint64_t * gpu_sums;
	hipMalloc(&gpu_sums, generateThreadCount * sizeof(uint64_t));
	uint64_t * gpu_totals;
	hipMalloc(&gpu_totals, reduceThreadCount * sizeof(uint64_t));

	int threadsPerBlock = 256;
	int blocks = (generateThreadCount + threadsPerBlock - 1) / threadsPerBlock;
	generatePoints<<<blocks, threadsPerBlock>>>(gpu_sums, generateThreadCount, sampleSize);

	// immediately launch reduce kernel as it uses data already on the gpu
	blocks = (reduceThreadCount + threadsPerBlock - 1) / threadsPerBlock;
	reduceCounts<<<blocks, threadsPerBlock>>>(gpu_sums, gpu_totals, generateThreadCount, reduceSize);

	hipMemcpy(sums, gpu_sums, generateThreadCount * sizeof(uint64_t), hipMemcpyDeviceToHost);
	hipMemcpy(totals, gpu_totals, reduceThreadCount * sizeof(uint64_t), hipMemcpyDeviceToHost);

	hipFree(gpu_sums);
	hipFree(gpu_totals);

	uint64_t hits = 0;
	for(i = 0; i < reduceThreadCount; i ++) hits += totals[i];
	free(totals);
	free(sums);

	approxPi = (double)(hits) / (double)(generateThreadCount * sampleSize) * 4;

	return approxPi;
}
